#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <math.h>
#include <omp.h>
#include <time.h>
#include <sys/types.h>
#include <unistd.h>
#include <fenv.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
 #include ""
//~ #include <hip/hip_runtime_api.h>
#include "../headers/init_coe.h"
#include "../headers/poinit_coe.h"
#warning "Compilation Error"
//~ #include "../headers/cu_init_coe.h"
#include "../headers/cu_poinit_coe.h"
//Hacer la proyección primero y aplicarla a cada píxel y no hacer la proyección para cada píxel.

//Crear  un array cant con tamaño NumPix*NumAng

//Inicializar coe's con CUDA

// for(NumAng)
// for (NPix)
// for(dimtotal)
//~ {
	
	//~ Suma_proyeccione
	
//~ }

// for(dimtotal)
//~ {
	
	//~ update_pixel
	
//~ }

#define PATH "/home/pablo/local.git/pablo"
#define CUDA 1

#define pi 3.141592
#define DOSPI
//~ #define COEF_COM
#define GUARDARC
#define PARALLEL_BEAM // =! RAND_ANG O SI?
#define DIMX 200
#define DIMY 200
#define FDIM 1.2
#define NANG 30
#define NPIX 600 //Esto sirve para algo ¿?
#define LIMX (int)(DIMX*FDIM)
#define LIMY (int)(DIMY*FDIM)
#define LIMTOT (int) (LIMX*LIMY)
//~ #define TX 10
//~ #define TY 20
//~ #define LIMTOT 20

#define D2N (int)(DIMX*DIMY*NANG)

//Error checking

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//~ #define D2N 40
#define ind_coe(a, b, c) ((a*DIMY+b)*NANG +c)  //Aqui da problemas, tienes que pensar bien cómo hacerlo
#define ind_coe2(a,b) (a*LIMY+b) //Para n_gamma
#define ind_coe3(a,b) (a*LIMTOT+b) //Para coe y para gamma al parecer
#define coe_gam(a,b) (a*LIMTOT+b)
void *carga_archivo(int NumAng, int NumPix, float *b);
FILE *apertura_archivo(int contador);
FILE *apertura_archivo2(int cont1, int cont2);
void saltar_com(FILE *fin);

 //~ typedef struct dim3{
	
	//~ int x;
	//~ int y;
//~ }

//~ typedef struct nopze{
	//~ int nonnum;
	//~ size_t size;
	//~ int ang[];
	
//~ }nopze;

//~ struct nopze2{
	//~ int nonnum;
	//~ int ang[20];
	
//~ };

//~ struct sparse_mat{
	
	//~ float *nozero;
	//~ int *space;
	
//~ }; 
//~ inline
//~ nopze* alloc_nopze(int a, size_t n) {
  //~ nopze * ret = calloc(sizeof(nopze) + n, 1);
  //~ if (ret) memcpy(ret,
                  //~ &(nopze const){ .nonnum=a, .size= n},
                  //~ sizeof(nopze));
  //~ return ret;
//~ } // a = 0; n = NumPix

int main(int argc, char** argv)
{	
	
	int NumPix, gammcon, cont, cont2, cont3, cont4, cont5, cont6, cont7, dimx, dimy, limx, limy, NumAng, facx, facxt, facy, facyt, i, valu, NumRayos1, ng, dimtotal, nozcoe, suma, ac1, ac2, c1, c2, gc1, gc2, bc1,  d2;
	int aint[3];
	float  factora, deltafantes, deltaang, lambda, deltaf, diferencia, cant, sumpix, factorpos, lol;
	char filename[100];
	FILE *finp, *pipeplot;
	clock_t inicio, final, com, fin;
	time_t archtime;
	float factordim = FDIM;
	int TX=32;
	int TY=32;
	int TXX=25;
	printf("Limx = %d, %d", LIMX, ind_coe(2,3,1));
	//~ getchar();
	//~ feenableexcept(FE_DIVBYZERO| FE_INVALID| FE_OVERFLOW);
	//~ exit(1);
	#ifdef DIM_TXT
	finp=fopen("../FP/input/dimSART.txt", "r");
	fscanf(finp, "%d". &dimx);
	fscanf(finp, "%d", &dimy);
	fclose(finp);
	#endif
	
	dimx=DIMX;
	dimy=DIMY;
	d2=D2N;
	lambda=0.01;
	finp=fopen("../FP/input/inputint.txt", "r");
	i=0; // ¿?
	for(cont=0; cont<3; cont++)
		{
			
			saltar_com(finp);
			fscanf(finp, "%d", &aint[cont]);
			printf("%d\n", aint[cont]);
			saltar_com(finp);
		}
	fclose(finp);
	printf("%d x %d\n", dimx, dimy);
	#ifdef DOSPI
	NumAng=aint[0];
	#endif
	
	#ifdef ABANICO
	//~ double angulos[10];
	finp=fopen("../FP/input/NumAngAbanico.txt", "r");
	fscanf(finp, "%d", &NumAng);
	fclose(finp);
	float *angulos=(float*)calloc(NumAng, sizeof(float));
	finp=fopen("../FP/input/AngulosBundle.txt", "r");
	for(cont=0; cont<NumAng; cont++)
	fscanf(finp, "%f", &angulos[cont]);
	fclose(finp);
	#endif

	NumPix=aint[2];
	
	printf("Antes de la declaracion de structs\n");
	//~ NumAng = aint;

	//~ NumRayos=NANG*NumPix;
	NumRayos1=(NANG+1)*NumPix;
	//~ limx=(int)rint(factordim*dimx);
	//~ limy=(int)rint(factordim*dimy);
	limx=LIMX;
	limy=LIMY;
	dimtotal=LIMTOT; 
    facx=(int)rint((factordim-1)*dimx/2);
	facy=(int)rint((factordim-1)*dimy/2);
	facxt=(int)rint((factordim+1)*dimx/2);
	facyt=(int)rint((factordim+1)*dimx/2);
	factorpos=(float)dimy/NumPix; 
	deltaang=2*pi/NumAng;
	printf("limx %d limy %d facs %d %d, LIMTOT %d y D2N %d, NumAng = %d\n", limx, limy, facx, facxt, LIMTOT, D2N, NumAng);
	size_t tamdimtotal = dimtotal*sizeof(float);
	  //~ exit(1);
	#ifdef DETORNOT
	if(NumRayos<(dimx*dimy))
		{
			printf("Undetermined system\n");
			return -1;
		}
    #endif
			//~ exit(1);
	//~ nopze n_gamm[LIMTOT];
	int no_gam[LIMTOT];
	//~ int no_gama[LIMTOT*NumPix];
	
	int *no_gama=(int*) calloc(LIMTOT*NumPix, sizeof(int));
	for(cont=0; cont<LIMTOT; cont++)
	{
		no_gam[cont]=0;
		
	}
	//~ for(cont=0; cont<LIMTOT; cont++)
	//~ {
		
		//~ n_gamm[cont]=alloc_nopze(0, NumPix);
		
	//~ }

	//~ nopze *a=alloc_nopze(0, NumPix);

	//~ nopze nzcoe[D2N]; //ind_coe definido para este siempre y cuando a vaya de 0 a dimx*dimy y b vaya de 0 a NAng
	int nzcoe[D2N];
	int *nzcoea=(int *)calloc(NumPix*d2, sizeof(int)); //runtime variable ¿?¿?¿?¿?¿?¿? Poss lost valgrind
	float *gamma=(float *)calloc(NumAng*dimtotal, sizeof(float)); //valgrind echoes error
	float *b=(float *)calloc(NumAng*NumPix,sizeof(float)); //Si const b, quitar esto. const float *b=read_input_data(); ¿?
	float *betar=(float *)calloc(NumAng*NumPix, sizeof(float)); 
	float *coe=(float *)malloc(dimtotal*NumRayos1*sizeof(float)); //valgrind error
	float *x = (float *) calloc(dimtotal, sizeof(float)); //Def lost
	float *loq=(float *) calloc(NumAng*NumPix, sizeof(float));
	float *difgama=(float *)calloc(dimtotal, sizeof(float));
	//~ #pragma omp parallel
	//~ {
			//~ #pragma omp for private(cont, cont2)
			//~ for(cont=0; cont<limx;cont++)
			//~ {
				
				//~ for(cont2=0; cont2<limy; cont2++)
				//~ {
					
					
						//~ n_gamm[ind_coe2(cont, cont2)].nonnum=0;
						//~ n_gamm[ind_coe2(cont, cont2)].ang=calloc(NANG, sizeof(int)); //valgrind error
					
				//~ }
			
			
			//~ }
		
		
		
	//~ }
	//~ #pragma omp parallel
	//~ {
		//~ #pragma omp for private(cont, cont2, cont3)
		//~ for(cont=0; cont<dimx; cont++)
		//~ {
			
			//~ for(cont2=0; cont2<dimy; cont2++)
			//~ {
				
				//~ for(cont3=0; cont3<NumAng; cont3++)
				//~ {
					
					//~ nzcoe[ind_coe(cont, cont2, cont3)].ang=calloc(NumPix, sizeof(int));
										
				//~ }				
				
			//~ }
								
		//~ }				
		
	//~ }
	
	#pragma omp parallel
	{
		
		#pragma omp for private(cont, cont2, finp, valu)
			for(cont=0; cont<NumAng; cont++)
			{
				//Nada claro esto de aqui abajo
				//~ gamma[cont]=calloc(dimtotal, sizeof(gamma[0]));
				//~ betar[cont]=calloc(NumPix, sizeof(betar[cont]));
				//~ b[cont]=calloc(NumPix,sizeof(b[0]));
				finp=apertura_archivo(cont);
				
				
				for(cont2=0; cont2<NumPix; cont2++)
				{
					
					if(cont2==0)
					b[cont*NumPix+cont2]=log(1+valu)+log(2); //No hace mucha diferencia
					//~ b[cont*NumPix+cont2])log(1+valu);
					else
					{
						
						fscanf(finp, "%d", &valu);
						b[cont*NumPix+cont2]=log(1+valu);
						
					}
					
					
				}
				
				fclose(finp);
			}
				
		
	}
	
	for(cont=0; cont<dimtotal; cont++)
	{
		x[cont]=0.f;
		difgama[cont]=0.f;
	}
	//CUDA inits Antes estaba dentro CHANGE 22-9
	float *coe_out;	
	hipMalloc(&coe_out, dimtotal*NumRayos1*sizeof(float));
	gpuErrchk( hipPeekAtLastError() );
	dim3 blockSize2(TXX, TXX);
	int axx=(dimtotal+TXX-1)/(TXX);
	int bxx=(NumRayos1+TXX-1)/TXX;
	dim3 gridSize2(axx, bxx);
	printf("Gridsize = %d, blockSize = %d\n", axx,TXX);
	initialize_coe<<<gridSize2, blockSize2>>>(coe_out, NumRayos1); //El segundo valor está bien puesto ¿?
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk(hipDeviceSynchronize());
	getchar();
	dim3 blockSize(TX, TY);
	int bx=(NumPix+TX-1)/TX; // (NumAng +1)
	int by=(NumAng+TY)/TY; //Antes ponia NumPix
	printf("bx = %d, by=%d\n", bx, by);
	//~ getchar();
	dim3 gridSize(bx,by);
	switch(CUDA){
		
		case(0):
	#ifdef PARALLEL_BEAM
	
	#ifdef DOSPI
	
	#pragma omp parallel 
	{
		
		#pragma omp for 
			for(cont=0; cont < NumAng; cont++)
			{
				//Se entra aqui de momento
				pa_initialize_coef_flo_2pi(cont, NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos,  &coe[0]);

			}
		
	}
	break;
	#endif
	
	#ifdef ABANICO
	
	#pragma omp parallel 
	{
		
			#pragma omp for 
			for(cont=0; cont < NumAng; cont++)  //Este for tiene sentido?
			{
				pa_initialize_coef_flo_abanico(cont, NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos, &angulos[0], &coe[0]);
			}
		
	}
	
	break;
	#endif
	
	
	
	#endif
	


	
	
	#ifdef POINT_SOURCE
	
	#ifdef DOSPI
	
	#pragma omp parallel 
	{
		
		#pragma omp for 
			for(cont=0; cont < NumAng; cont++)
			{
				
				po_initialize_coef_flo_2pi(cont, NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos,  coe);
			}
		
	}
	break;
	#endif
	
	#ifdef ABANICO
	
	#pragma omp parallel 
	{
		
		 	#pragma omp for 
			for(cont=0; cont < NumAng; cont++)
			{
				po_initialize_coef_flo_abanico(cont, NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos, angulos, coe);
			}
		
	}
	
	break;
	#endif
	
	
	
	#endif
	
	case(1):
	
	//~ hipMalloc((void**)&coe_out, dimtotal*NumRayos1*sizeof(float));

	//~ dim3 blockSize2(TXX);
	//~ int axx=((dimtotal*NumRayos1)+TXX-1)/TXX;
	//~ dim3 gridSize2(axx);
	//~ initialize_coe<<<gridSize2, blockSize2>>>(coe_out);
	//~ hipDeviceSynchronize();
	//~ dim3 blockSize(TX, TY);
	//~ int bx=(NumAng+TX)/TX; // (NumAng +1)
	//~ int by=(NumPix+TY-1)/TY; //Antes ponia NumPix
	//~ printf("bx = %d, by=%d\n", bx, by);
	//~ getchar();
	//~ dim3 gridSize(bx,by);
	//~ gpuErrchk(hipMemcpy(coe_out, coe, dimtotal*NumRayos1*sizeof(float), hipMemcpyHostToDevice)); //CHANGE 22-9 CHANGE 28-9 Lo he quitado, no sabía qué pintaba aquí
	#ifdef PARALLEL_BEAM
	
	#ifdef DOSPI
	cu_pa_initialize_coef_flo_2pi<<<gridSize, blockSize>>>(NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos, coe_out);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );
	//~ cu_pa_initialize_coef_flo_2pi<<<(1,1), (NumAng, NumPix)>>>(NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos, coe_out);
	//~ hipDeviceSynchronize();
	printf("Salio del Kernel\n");

	hipMemcpy(coe, coe_out, dimtotal*NumRayos1*sizeof(float), hipMemcpyDeviceToHost);	
	
	for(i=0; i<(dimtotal); i++)
	printf("coe_out[%d] = %f\n", i, coe[i]);
	getchar();
	hipFree(coe_out);
	break;
	#endif
	
	#ifdef ABANICO
	cu_pa_initialize_coef_flo_abanico<<<gridSize, blockSize>>>(NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, factorpos, angulos, coe_out);
	hipMemcpy(coe, coe_out, dimtotal*NumRayos1*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(coe_out);
	break;
	#endif
	#endif
	#ifdef POINT_SOURCE
	
	
	
	#ifdef DOSPI
	cu_po_initialize_coef_flo_2pi<<<gridSize, blockSize>>>(NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, deltaang, factorpos, coe_out);
	hipMemcpy(coe, coe_out, dimtotal*NumRayos1*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(coe_out);
	break;
	#endif
	
	#ifdef ABANICO
	cu_po_initialize_coef_flo_abanico<<<gridSize, blockSize>>>(NumPix, NumAng, factordim, limy, dimx, dimy, dimtotal, NumRayos1, factorpos, angulos, coe_out);
	hipMemcpy(coe, coe_out, dimtotal*NumRayos1*sizeof(float), hipMemcpyDeviceToHost);
	hipFree(coe_out);
	break;
	#endif
	
	#endif
	
}
	
	finp=fopen("../res/Suma_Coef_SART.txt", "w+");
	for(cont=0;	cont<limx; cont++)
	{
		
		for(cont2=0; cont2<limy; cont2++){
			c1=NumAng*NumPix;
			c2=cont*limy+cont2;	
			fprintf(finp, "%d %d %f\n", cont, cont2, coe[ind_coe3(c1, c2)]);
		}
	}
	fclose(finp);
	
	//~ #pragma omp parallel
		//~ {
			
			//~ #pragma omp for private(cont, finp, cont2, cont3, cont4)
				//~ for(cont=0; cont<NumAng; cont++)
					//~ {
						//~ for(cont2=0; cont2<NumPix; cont2++)
							//~ {
								
								//~ finp=apertura_archivo2(cont, cont2);
									//~ for(cont3=0; cont3<limx; cont3++)
										//~ {
											
											//~ for(cont4=0; cont4<limy; cont4++)
												//~ fprintf(finp, "%d %d %f\n", cont3, cont4, coe[cont*NumPix+cont2][cont3*limy+cont4]);
											
											
										//~ }
								
								
							//~ }
						
						
						
					//~ }

			
			
		//~ }
		
	for(cont=0; cont<NumAng; cont++)
	{
		for(cont2=0; cont2<NumPix; cont2++)
		{
			
			for(cont3=facx; cont3<facxt; cont3++)
			{
				for(cont4=facy; cont4<facyt; cont4++)
				{
					c1=cont*NumPix+cont2;
					c2=cont3*limy+cont4;
					betar[cont*NumPix+cont2]+= (coe[ind_coe3(c1,c2)]);
				}
				
				
			}
			
		}
		
		
	}
		
	gammcon=0;	
		
	for(cont=0; cont<NumAng; cont++)
	{
		
		for(cont2=facx; cont2<facxt; cont2++)
		{
			for(cont3=facy; cont3<facyt; cont3++)
			{
				for(cont4=0; cont4<NumPix; cont4++)
				{
			
					c1=cont*NumPix+cont4;
					c2=cont2*limy+cont3;
					gc1=cont;
					gc2=cont2*limy+cont3;
					gamma[ind_coe3(gc1,gc2)]+=(coe[ind_coe3(c1,c2)]);
					
				}
				if(gamma[ind_coe3(gc1,gc2)]<0.0001)
					gammcon++;
			}
			
			
			
		}
		
	}
		
	i=0;
    deltaf=1;
	//~ finp=fopen("../res/debug_data.txt", "at");
	//~ fprintf(finp, "%f %f %f %f %f gammcon %d\n", coe[ind_coe3(3000,445)], b[20*NumPix+100], betar[2*NumPix+2], gamma[ind_coe3(0,665)], gamma[ind_coe3(0,666)], gammcon);
	//~ fclose(finp);
	//~ for(cont=0; cont<limx; cont++)
	//~ {
		//~ for(cont2=0; cont2<limy; cont2++)
			//~ x[cont*limy+cont2]=0.f;
	//~ }
	printf("Dim %d, resta %d\n", dimx, (facxt-facx));
	for(cont2=facx; cont2<facxt; cont2++)
	{
		
		for(cont3=facy; cont3<facyt; cont3++)
		{
			
			for(cont=0; cont<NumAng; cont++)
			{
				nozcoe=0;  
					for(cont4=0; cont4<NumPix; cont4++)
					{	c1=cont*NumPix+cont4;
						c2=cont2*limy+cont3;
						if(coe[ind_coe3(c1,c2)]>0.f)
						{
							//~ nzcoe[(cont2-facx)*dimy+(cont3-facy)][cont].ang[nozcoe]=cont4;
							nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+nozcoe]=cont4;
							nozcoe++;
							
						}
						
					}
				
				nzcoe[ind_coe((cont2-facx),(cont3-facy),cont)]=nozcoe;
				
			}
		
			
			
		}
		
		
	}
	suma=0;
	for(cont2=facx; cont2<facxt; cont2++)
	{
		for(cont3=facy; cont3<facyt; cont3++)
		{
				for(cont4=0; cont4<NumPix; cont4++)
					suma+=nzcoe[ind_coe((cont2-facx),(cont3-facy),cont)];
		}


	}
        printf("Sparcity = %d de %d\n", suma, NumAng*NumPix*dimx*dimy);
        
//Recorre las gamma (suma de las columnas de la matriz de coef) buscando las no nulas
	for(cont2=facx; cont2<facxt; cont2++)
	{
	
		for(cont3=facy; cont3<facyt; cont3++)
		{

				ng=0;
				for(cont=0; cont<NumAng; cont++)
				{		gc2=cont2*limy+cont3;
						if(gamma[ind_coe3(cont, gc2)] > 0.f)
						{
							no_gama[ind_coe2(cont2, cont3)*NumPix+ng]=cont;
							ng++;
								//~ printf("Se cumple lo de gamma");
						}
				}
				no_gam[ind_coe2(cont2, cont3)]=ng;

		}
	
	}
	betar[0]=0.01;
	inicio=clock();
	deltafantes=100; deltaf=10;
	printf("Comienza iter\n");
	pipeplot=popen("gnuplot -persist","w");
	fprintf(pipeplot, "set palette gray negative\n");
	
	//while((deltaf>0.1) && (i<50) && (deltafantes>deltaf))
	//{
		//deltafantes=deltaf;
		//deltaf=0;
		//sumpix=0;
		
		//for(cont2=facx; cont2<facyt; cont2++)
		//{
			
			//for(cont3=facy; cont3<facyt; cont3++)
			//{
				
				//difgam=0;
					////~ com=clock();
				//for(cont=0; cont<no_gam[ind_coe2(cont2, cont3)]; cont++)
				//{
					//diferencia=0;
					
					//#pragma omp parallel for private(cont4, cant, cont5, cont6, bc1, bc2, c1, c2, c3, lol, ac1, ac2) reduction(+:diferencia) shared(coe)
					
					//for(cont4=0; cont4<nzcoe[ind_coe((cont2-facx),(cont3-facy),cont)]; cont4++)
					//{
						
						//cant=0;
						//c1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont]*NumPix+nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
						//c2=cont2*limy+cont3;
						//lol=coe[ind_coe3(c1,c2)];
						
						//if(lol>0.f){
							
					
						
						////~ printf("Dentro bucle y cont = %d\n", cont);
						
						
						
							////~ printf("Pro es %d\n", pro);
							////~ getchar();
					
						//for(cont5=facx; cont5<facxt; cont5++)
						//{
							
							//for(cont6=facy; cont6<facyt; cont6++)
							//{
								//ac1=cont*NumPix +nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
								//ac2=cont5*limy+cont6;
								
								//if(coe[ind_coe3(ac1,ac2)]>0)
								//cant+=coe[ind_coe3(ac1,ac2)]*x[ac2]; //Aquí sparse_matrix_multip ¿?
								
								
							//}
							
							
						//}
		
						
						
						//bc1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont];
						//bc2=nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
						//bc1=bc1*NumPix+bc2;
						//c3=no_gama[ind_coe2(cont2, cont3)*NumPix+cont]*NumPix+nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
						//diferencia+=(b[c3]-cant)*(coe[ind_coe3(c1,c2)])/(betar[bc1]);  //Calculo de la diferencia para cada rayo entre la proyección y lo calculado
						
					//}
					
					//}
					
					
					//gc1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont];
					//gc2=cont2*limy+cont3;
					//difgam+=diferencia/(gamma[ind_coe3(gc1, gc2)]); //Esto se podría hacer antes... Se calcula la suma de las diferencias para cada rayo
				//}
				////~ fin=clock();
				////~ printf("Update de pixel tarda %lf\n",(double)(fin-com)/(CLOCKS_PER_SEC));
				////~ getchar();
				//x[cont2*limy+cont3]+=lambda*difgam; //update del pixel
				//deltaf+=lambda*difgam; //variables de control de la convergencia
				//sumpix+=x[cont2*limy+cont3]; // ¿?
				 
			//}  //Final bucle update de un pixel
			
			
			
		//}
		
		//deltaf=deltaf/sumpix;
		//deltaf=fabs(deltaf);
		//final=clock();
		//printf("Deltaf=%lf y n de iteraciones =%d. Desde el inicio llevamos %lf segundos\n", deltaf, i,(double)(final-inicio)/(CLOCKS_PER_SEC*12));
        //sprintf(filename, "../res/Resultados_SART/Resultados_SART_%d.txt", i);

		//finp=fopen(filename, "w+");
		
		//for(cont=0; cont<limx; cont++)
		//{
			//for(cont2=limy; cont2>0; cont2--)
				//fprintf(finp, "%d %d %f\n", cont, limy-cont2, x[cont*limy+cont2]);
			
		//}
		
		//fclose(finp);

        //fprintf(pipeplot, "plot '%s/res/Resultados_SART/Resultados_SART_%d.txt'w image \n",PATH, i);
        //fprintf(pipeplot, "set term postscript \n");

        //fprintf(pipeplot, "set output 'img/imSART/NumAng_%d_Numpix_%d,dim_%dx%d__%d.png' \n", NumAng, NumPix, dimx, dimy, archtime);
        //fprintf(pipeplot, "replot\n");
        //fflush(pipeplot);
        //i+=1;

	//}


while((deltaf>0.01) && (i<100)&& (deltafantes>deltaf) ) //
	{
		deltafantes=deltaf;
		deltaf=0;
		sumpix=0;
		
	//~ com=clock();
				for(cont=0; cont<NumAng; cont++)
				{
					diferencia=0;
					
					#pragma omp parallel for private(cont4, cant, cont5, cont6, bc1,  c1, c2,  lol, ac1, ac2) reduction(+:diferencia) shared(coe)
					
					for(cont4=0; cont4<NumPix; cont4++)
					{
					
					
						
						
						
						
						
							//~ printf("Pro es %d\n", pro);
							//~ getchar();
					
						for(cont5=facx; cont5<facxt; cont5++)
						{
							
							for(cont6=facy; cont6<facyt; cont6++)
							{
								ac1=cont*NumPix+cont4;
								ac2=cont5*limy+cont6;
								
								if(coe[ind_coe3(ac1,ac2)]>0)
								loq[ac1]+=coe[ind_coe3(ac1,ac2)]*x[ac2]; //Aquí sparse_matrix_multip ¿?
								
								
							}
							
							
							
							
						}
						
					
					
				}
				
			}
			cont7=0;
			
			//Calculo de factor de corrección para cada pixel
			for(cont2=facx; cont2<facyt; cont2++)
			{
					for(cont3=facy; cont3<facyt; cont3++)
					{	c2=cont2*limy+cont3;
							for(cont=0; cont<no_gam[ind_coe2(cont2, cont3)]; cont++)
							{
								diferencia=0;
								bc1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont];
								
								
								for(cont4=0; cont4<nzcoe[ind_coe((cont2-facx),(cont3-facy),cont)]; cont4++)
								{
									
									c1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont]*NumPix+nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
								
									lol=coe[ind_coe3(c1,c2)];
									
									
									//~ bc2=nzcoea[ind_coe((cont2-facx),(cont3-facy),cont)*NumPix+cont4];
									bc1++;

									diferencia+=(b[c1]-loq[bc1])*lol/(betar[bc1]); 
									
								
					
								}
					
					
						gc1=no_gama[ind_coe2(cont2, cont3)*NumPix+cont];
						gc2=cont2*limy+cont3;
						difgama[gc2]+=diferencia/(gamma[ind_coe3(gc1, gc2)]);
						}
						
						//~ printf("Hecho para pixel %d\n", cont7);
						cont7++;
					}
				
			}
			
			
			cont7=0;
			
			//update de cada pixel
		for(cont2=facx; cont2<facyt; cont2++)
		{
			
			for(cont3=facy; cont3<facyt; cont3++)
			{
				
				

				//~ printf("Update de pixel %d de %d tarda %lf y es %f\n", cont7, (facyt-facy)*(facxt-facx), (double)(fin-com)/(CLOCKS_PER_SEC), lambda*difgama[cont2*limy+cont3]);
				cont7++;
				//~ getchar();
				x[cont2*limy+cont3]+=lambda*difgama[cont2*limy+cont3];
				deltaf+=lambda*difgama[cont2*limy+cont3];
				sumpix+=x[cont2*limy+cont3];
				 
			}  //Final bucle update de un pixel
			
			
			
		}
		deltaf=deltaf/sumpix;
		deltaf=fabs(deltaf);
		final=clock();
		printf("Deltaf=%lf y n de iteraciones =%d. Desde el inicio llevamos %lf segundos\n", deltaf, i,(double)(final-inicio)/(CLOCKS_PER_SEC*12));
		//~ getchar();
        sprintf(filename, "../res/Resultados_SART/Resultados_SART_%d.txt", i);

		finp=fopen(filename, "w+");
		
		for(cont=0; cont<limx; cont++)
		{
			for(cont2=limy; cont2>0; cont2--)
			
			
				fprintf(finp, "%d %d %f\n", cont, limy-cont2, x[cont*limy+cont2]);  
			
		}
		
		fclose(finp);

       
        i+=1;

	} 
	
	fprintf(pipeplot, "plot '%s/res/Resultados_SART/Resultados_SART_%d.txt'w image \n",PATH, (i-1));
        fprintf(pipeplot, "set term postscript \n");

        fprintf(pipeplot, "set output 'img/imSART/NumAng_%d_Numpix_%d,dim_%dx%d__%d.png' \n", NumAng, NumPix, dimx, dimy, archtime);
        fprintf(pipeplot, "replot\n");
       
	fflush(pipeplot);
	pclose(pipeplot);
	
	#ifdef GUARDARC
	
	sprintf(filename, "../res/SARTC/NAng_%d.txt", NumAng);
	finp=fopen(filename, "w+");
	
	 for(cont=0; cont<limy; cont++)
	{

		for(cont2=0; cont2<limx; cont2++)
		{
			fprintf(finp, "%d %d %f\n",cont, cont2, x[cont*limx+cont2]);
		}

	}

	fclose(finp);
	
	#endif
	
	//~ #pragma omp parallel
	//~ {
		
		//~ #pragma omp for private(cont)
		//~ for(cont=0; cont<NumRayos1; cont++)
		//~ {
			
			//~ free(coe[cont]); //valgrind error
			
		//~ }
				
	//~ }
	
		//~ #pragma omp parallel
	//~ {
		//~ #pragma omp for private(cont, cont2, cont3)
		//~ for(cont=0; cont<dimx; cont++)
		//~ {
			
			//~ for(cont2=0; cont2<dimy; cont2++)
			//~ {
				
				//~ for(cont3=0; cont3<NumAng; cont3++)
				//~ {
					
					//~ free(nzcoe[ind_coe(cont, cont2, cont3)].ang);
										
				//~ }				
				
			//~ }
								
		//~ }				
		
	//~ }
	
	
	//~ #pragma omp parallel
	//~ {
			//~ #pragma omp for private(cont, cont2)
			//~ for(cont=0; cont<limx;cont++)
			//~ {
				
				//~ for(cont2=0; cont2<limy; cont2++)
				//~ {
					
					
					
						//~ free(n_gamm[ind_coe2(cont, cont2)].ang);
					
				//~ }
			
			
			//~ }
		
		
		
	//~ }
	//~ free(no_gam);
	free(no_gama);
	free(gamma);
	free(difgama);
	free(coe);
	free(b);
	free(nzcoea);
	free(x);
	free(betar);
	free(loq);
	//~ for(cont=0; cont<NANG; cont++)
	//~ free(gamma[cont]);
	
	return 0;
}



void *carga_archivo(int NumAng, int NumPix, float *b) /*Esta función está en el main tal cual*/
{
        int cont, cont2, valu;

        FILE *finp;

 for(cont=0; cont<NumAng; cont++)
 {

         finp=apertura_archivo(cont);

         for(cont2=0; cont2<NumPix; cont2++)
         {
        fscanf(finp, "%d", &valu);

        b[cont*NumPix+cont2]=log(1+valu); /*Esto aquí lo he cambiado*/
        //~ b[cont][cont2]=valu;
        if(cont2==0)
        b[cont*NumPix+cont2]=log(2)+log(1+valu);

         }

         fclose(finp);
 }

        //~ printf("Archivo cargado, presione alguna tecla\n");
        //~ getchar();
}


FILE *apertura_archivo(int contador)
{

char nombrearchivo[250]="Resultados_pixeles.txt";

sprintf(nombrearchivo, "%s/res/angulo_%d.txt",PATH, contador);

if(fopen(nombrearchivo, "r")==NULL)
{
        perror("El archivo no existe o no se puede abrir\n");
}

return fopen(nombrearchivo, "r");



}

                                                              
void saltar_com(FILE *fin) /* En archivo de texto salta a la siguiente línea tras \n*/
{
        char col;
        while(fscanf(fin, "%c", &col))
        {
                if(col=='\n')
                {

                        break;
                }

        }


}

FILE *apertura_archivo2(int cont1, int cont2)
{

        char file_name[200];
        sprintf(file_name, "%s/res/Resultados_SART/Matriz_coef_SART_%d_%d.txt", PATH, cont1, cont2);


        return fopen(file_name, "w+");
}


